
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
__global__ void add2_kernel(float* c,
                            const float* a,
                            const float* b,
                            int n) {
    //printf("i start: %d \n ",blockIdx.x * blockDim.x + threadIdx.x);

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
            i < n; i += gridDim.x * blockDim.x) {
        c[i] = a[i] + b[i];
        //printf("idx: %d , val a: %f: \n ",i,a[i]);		
    }
}

void launch_add2(float* c,
                 const float* a,
                 const float* b,
                 int n) {
    dim3 grid((n + 1023) / 1024);
    dim3 block(1024);

    hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start, 0); // Record an event in the default 


    add2_kernel<<<grid, block>>>(c, a, b, n);

hipEventRecord(stop, 0); // Record another event after the kernel launch
hipEventSynchronize(stop); // Wait for the event to be recorded
float elapsedTime;
hipEventElapsedTime(&elapsedTime, start, stop);
std::cout << "Elapsed time: " << elapsedTime << " ms." << std::endl;
hipEventDestroy(start);
hipEventDestroy(stop);

}

// int main(void){
	
// 	float a[3]={1,2,3};
// 	float b[3]={4,5,6};
// 	float c[3]={0,0,0};
// 	launch_add2(a,b,c,3);


	
// return 0;

// }
